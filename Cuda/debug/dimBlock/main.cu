/*
This code is show errors in Cuda code:
1. The maximum nubmer of threads in a block is 1024, so if you set dimBlock to be dimBlock(64,64,1), you will see an error: 
  invalid configuration argument.

2.If the configuration is correct, when you run, you see another error:
 an illegal memory access was encountered.

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void foo(int *ptr) 
{
  *ptr =7;

}

int main() {

//dim3 dimBlock(64,64,1);
dim3 dimBlock(32,32,1);
foo<<<1,dimBlock>>>(0);

hipDeviceSynchronize();
hipError_t error = hipGetLastError();
if (error !=hipSuccess) {
  printf("CUDA error: %s\n", hipGetErrorString(error));
		exit(-1);
}

return 0;
}
