#include "hip/hip_runtime.h"
//When you set the numthreadsBlock = 64 or smaller, you can not get the corret answer; still the index is not correct
// when testing the code, k should be an odd number because k is assumed to be odd when computing the truth
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void max_each_block(float *dmaxarr, int *dmaxstart, int *dmaxend, float * darr,  int n, int k);
void maxburst(float *x, int n, int k, int *startend, float *bigmax);
void find_max_from_blocks(float *maxarr, int *maxstart, int *maxend, int numBlock,int *startend, float *bigmax);

int main(int argc, char **argv) {
				int n = atoi(argv[1]);
				int k = atoi(argv[2]); 
				
		//generate a 1d array
				float *arr = (float*) malloc(n*sizeof(float));
				int i;
				for (i = n; i > 0; i--) {
							arr[n-i] = (float)i;			
				}

			int startend[2];
			float bigmax;

   maxburst(arr, n,k, startend, &bigmax); 
	 	return 0;
}


__global__ void max_each_block(float *dmaxarr, int *dmaxstart, int *dmaxend, float * darr, int n,int k) {
 // declare three array for the maximum found by each thread 
  extern __shared__ float sh[];
  float *mymaxvals = sh;
  int *mystartmaxes = (int *)&mymaxvals[blockDim.x];
  int *myendmaxes = (int *)&mystartmaxes[blockDim.x];

  int perstart = threadIdx.x + blockDim.x * blockIdx.x;
  int perlen, perend;
  double xbar; // a temporay variable used when computing mean of subsequence

  int i, tid = threadIdx.x;

  if (perstart <= n-k) {
    for (perlen = k ; perlen <= n - perstart ; perlen++) {
      perend = perstart + perlen - 1;
      //compute the mean of subsequence incrementally
      if (perlen ==k) {
         xbar = 0;
         for ( i = perstart; i <= perend; i++) {
           xbar += darr[i];
         }
         xbar /= (perend - perstart + 1);
         mymaxvals[tid] = xbar;
         mystartmaxes[tid] = perstart;
         myendmaxes[tid] = perend;
      } else {
        xbar = ( (perlen-1) * xbar + darr[perend] ) / perlen;
      }
      //update the mymaxvals[tid] if the next subsequence in a thread has a higher mean
      if (xbar >  mymaxvals[tid]) {
         mymaxvals[tid] = xbar;
         mystartmaxes[tid] = perstart;
         myendmaxes[tid] = perend;
      }
    }
  } else {
    mymaxvals[tid] = 0;//initialize it with the smallest number
  }
  __syncthreads(); //sync to make sure each thread in this block has done with the for loop
//  // get the highest among the mymaxvals using reduce
    for (int s = blockDim.x/2; s > 0; s>>=1) {
     if (tid < s ) {
       if(mymaxvals[tid+s] > mymaxvals[tid]) {
          mymaxvals[tid] =  mymaxvals[tid+s];
          mystartmaxes[tid] = mystartmaxes[tid + s];
          myendmaxes[tid] = myendmaxes[tid + s];
       }
     }
     __syncthreads();
  }
  // the maximum among the mymaxvals in this block
 if(tid == 0) {
  dmaxarr[blockIdx.x] = mymaxvals[0];
  dmaxstart[blockIdx.x] =  mystartmaxes[0];
  dmaxend[blockIdx.x] =  myendmaxes[0];
  }
}

void maxburst(float *x, int n, int k, int *startend, float *bigmax) {
		const int numthreadsBlock = 1024;
		int numBlock = ( n + numthreadsBlock - 1)/numthreadsBlock;
		//declare arrays on cpu to store the results from the kernel
		float *maxarr = (float *)malloc(numBlock * sizeof(float));
  int *maxstart = (int *)malloc(numBlock * sizeof(int));
  int *maxend = (int *)malloc(numBlock * sizeof(int));

		// declare GPU memory pointers
		float *darr, * dmaxarr;
		int *dmaxstart, *dmaxend;
		hipMalloc((void **)&darr, n*sizeof(float));
		hipMalloc((void **)&dmaxarr, numBlock*sizeof(float));
		hipMalloc((void **)&dmaxstart, numBlock*sizeof(int));
		hipMalloc((void **)&dmaxend, numBlock*sizeof(int));

		//copy the input x to device
		hipMemcpy(darr, x, n*sizeof(float), hipMemcpyHostToDevice);
  // execution configuration
		dim3 dimGrid(numBlock,1);
		dim3 dimBlock(numthreadsBlock,1,1);
  //call the kernel
		max_each_block<<<dimGrid,dimBlock,(3*numthreadsBlock)*sizeof(float)>>>(dmaxarr,dmaxstart,dmaxend, darr, n, k);
		hipDeviceSynchronize();
		//copy the results from device to cpu
		hipMemcpy(maxarr, dmaxarr, numBlock*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(maxstart, dmaxstart, numBlock*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(maxend, dmaxend, numBlock*sizeof(int), hipMemcpyDeviceToHost);
  //free memory on device
		hipFree(darr);
		hipFree(dmaxarr);
		hipFree(dmaxstart);
		hipFree(dmaxend);

  find_max_from_blocks( maxarr, maxstart, maxend,  numBlock,startend, bigmax);
  printf("burst start from %d end at %d; max-mean is %f\n", startend[0], startend[1], *bigmax);

}

void find_max_from_blocks(float *maxarr, int *maxstart, int *maxend, int numBlock,int *startend, float *bigmax) {
	*bigmax = 0;
	for (int i = 0; i < numBlock; i++) {
   if (*bigmax < maxarr[i]) {
      *bigmax = maxarr[i];
      startend[0] = maxstart[i];
     	startend[1] = maxend[i];
			}	
 }
}
