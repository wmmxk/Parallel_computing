#include "hip/hip_runtime.h"
// when testing the code, k should be an odd number because k is assumed to be odd when computing the truth
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void parallel_max_each_chunk(float *dmaxarr, float * darr,  int n, int k);

int main(int argc, char **argv) {
				int n = atoi(argv[1]);
				int k = atoi(argv[2]); 
				
		//generate a 1d array
				float *arr = (float*) malloc(n*sizeof(float));
				int i;
				for (i = n; i > 0; i--) {
							arr[n-i] = (float)i;			
				}

		const int numthreadsBlock = 1024;
		int numChunk;
		numChunk = ( n + numthreadsBlock - 1)/numthreadsBlock;
		float *maxarr = (float *)malloc(numChunk * sizeof(float));

  int numBlock = numChunk;

		// declare GPU memory pointers
		float *darr, * dmaxarr;
		hipMalloc((void **)&darr, n*sizeof(float));
		hipMalloc((void **)&dmaxarr, numChunk*sizeof(float));
		hipMemcpy(darr, arr, n*sizeof(float), hipMemcpyHostToDevice);

		dim3 dimGrid(numBlock,1);
		dim3 dimBlock(numthreadsBlock,1,1);

		parallel_max_each_chunk<<<dimGrid,dimBlock,(3*numthreadsBlock)*sizeof(float)>>>(dmaxarr, darr, n, k);
		hipDeviceSynchronize();
		hipMemcpy(maxarr, dmaxarr, numChunk*sizeof(float), hipMemcpyDeviceToHost);

  //truth
				float *smaxarr = (float *)malloc(numChunk*sizeof(float));
				for (i = 0; i < numChunk; i ++) {
        smaxarr[i] = (i)*numthreadsBlock +k  <=n? arr[i*numthreadsBlock + k/2 ]:0; // k is an odd number
				}

		//check the results
				bool judge = true;
				for (i=0; i < numBlock; i++) {
						printf("max of block  %d,  %f %f\n ", i, smaxarr[i], maxarr[i]);
						judge = judge && (smaxarr[i] == maxarr[i]);
				}
				printf("\n--------correct or wrong---------\n");
				printf(judge ? "right\n": "wrong\n");

				// This is for developing: print out the 1d array

				printf("\n--------1d array---------\n");
    if ( n < 15) {
				for (i=0; i < n; i++) {
						printf("element  %d,  %f\n ", i, arr[i]);
				}
    }
		// check the exit state of CUDA code
		hipError_t error = hipGetLastError();
		if (error !=hipSuccess) {
				printf("CUDA error: %s\n", hipGetErrorString(error));
		}

		//free gpu memory
		hipFree(dmaxarr);
		hipFree(darr);
		return 0;
}

__global__ void parallel_max_each_chunk(float *dmaxarr, float * darr, int n,int k) {
 	int i, tid = threadIdx.x;

  // declare three array for the maximum found by each thread 
  extern __shared__ float mymaxvals[];
  extern __shared__ float mystartmaxes[];
  extern __shared__ float myendmaxes[];
		
		int perstart = threadIdx.x + blockDim.x * blockIdx.x;
		int perlen, perend;
		double xbar; // a temporay variable used when computing mean of subsequence

		if (perstart <= n-k) {
    for (perlen = k ; perlen <= n - perstart ; perlen++) {
				  perend = perstart + perlen - 1;
						//compute the mean of subsequence incrementally
      if (perlen ==k) {
							  xbar = 0;
									for ( i = perstart; i <= perend; i++) {
           xbar += darr[i];     
									}
									xbar /= (perend - perstart + 1);
									mymaxvals[tid] = xbar;
						} else {
        xbar = ( (perlen-1) * xbar + darr[perend] ) / perlen;
						}
						//update the mymaxvals[tid] if the next longer subsequence has a higher mean
						if (xbar > mymaxvals[tid]) {
         mymaxvals[tid] = xbar;
									mystartmaxes[tid] = perstart;
									myendmaxes[tid] = perend;
						}
				}
		} else {
    mymaxvals[tid] = 0;//initialize it the smallest number
		}
		__syncthreads(); //sync to make sure each thread in this block has done with the for loop

  // get the highest among the mymaxvals using reduce
  for (int s = blockDim.x/2; s > 0; s>>=1) {
     if (tid < s ) {
       if(mymaxvals[tid+s] > mymaxvals[tid]) {
						   	mymaxvals[tid] = 	mymaxvals[tid+s]; 
          mystartmaxes[tid] = mystartmaxes[tid + s];
									 myendmaxes[tid] = myendmaxes[tid + s];	
							}	
     }
     __syncthreads();
  }
  // the maximum among the mymaxvals in this block
  if(tid == 0) {
  dmaxarr[blockIdx.x] = mymaxvals[0];
  }
}

