#include "hip/hip_runtime.h"
/*  This code implements the serial solution and CUDA version for finding the maximal burst in a time series;

    How to compile:
        nvcc compare.cu 
    How to run:
        ./a.out n k //n is the length of the time series and k is the minimum lenght of a subsequence

    Results to see:
	       The burst found by two methods are printed out: "burst start from .. end at ..; max-me is .."			
    Notes:
        The serial takes long time for a large n (e.g n=3000) 
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

// kernel function
__global__ void max_each_block(float *dmaxarr, int *dmaxstart, int *dmaxend, float * darr,  int n, int k);

// max_burst calls the kernel, return three arrays, one for the maxval, one for startpoint, one for the endpoint in each block;
void max_burst(float *x, int n, int k, int *startend, float *bigmax);

//find the maximum from the returned arrays from the kernel. This function is called by max_burst
void find_max_from_blocks(float *maxarr, int *maxstart, int *maxend, int numBlock,int *startend, float *bigmax);

//serial solution
void s_max_burst(float *arr, int n, int k);
float mean(float *y, int s, int e);

int main(int argc, char **argv) {
				int n = atoi(argv[1]);
				int k = atoi(argv[2]); 
				
		 //generate a 1d array
				float *arr = (float*) malloc(n*sizeof(float));
				int i;
				for (i = n; i > 0; i--) {
							arr[n-i] = (float)(rand() % 80);			
				}

			// Cuda solution
			int startend[2];
			float bigmax;
   max_burst(arr, n,k, startend, &bigmax); 
		 // serial solution	
   s_max_burst(arr, n,k);

			return 0;
}

__global__ void max_each_block(float *dmaxarr, int *dmaxstart, int *dmaxend, float * darr, int n,int k) {
  // declare three array for the maximum found by each thread 
	 // learning material for shared memory: https://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
  extern __shared__ float sh[];
  float *mymaxvals = sh;
  int *mystartmaxes = (int *)&mymaxvals[blockDim.x];
  int *myendmaxes = (int *)&mystartmaxes[blockDim.x];

  int perstart = threadIdx.x + blockDim.x * blockIdx.x;
  int perlen, perend;
  double xbar; // a temporay variable used when computing mean of subsequence

  int i, tid = threadIdx.x;

  if (perstart <= n-k) {
    for (perlen = k ; perlen <= n - perstart ; perlen++) {
      perend = perstart + perlen - 1;
      //compute the mean of subsequence incrementally
      if (perlen ==k) {
         xbar = 0;
         for ( i = perstart; i <= perend; i++) {
           xbar += darr[i];
         }
         xbar /= (perend - perstart + 1);
         mymaxvals[tid] = xbar;
         mystartmaxes[tid] = perstart;
         myendmaxes[tid] = perend;
      } else {
        xbar = ( (perlen-1) * xbar + darr[perend] ) / perlen;
      }
      //update the mymaxvals[tid] if the next subsequence in a thread has a higher mean
      if (xbar >  mymaxvals[tid]) {
         mymaxvals[tid] = xbar;
         mystartmaxes[tid] = perstart;
         myendmaxes[tid] = perend;
      }
    }
  } else {
    mymaxvals[tid] = 0;//initialize it with the smallest number
  }
  __syncthreads(); //sync to make sure each thread in this block has done with the for loop
   // get the highest among the mymaxvals using reduce
			for (int s = blockDim.x/2; s > 0; s>>=1) {
			if (tid < s ) {
				if(mymaxvals[tid+s] > mymaxvals[tid]) {
							mymaxvals[tid] =  mymaxvals[tid+s];
							mystartmaxes[tid] = mystartmaxes[tid + s];
							myendmaxes[tid] = myendmaxes[tid + s];
				}
			}
			__syncthreads();
  }
 //put the maximum among the mymaxvals in this block to dmaxarr
 if(tid == 0) {
  dmaxarr[blockIdx.x] = mymaxvals[0];
  dmaxstart[blockIdx.x] =  mystartmaxes[0];
  dmaxend[blockIdx.x] =  myendmaxes[0];
  }
}

void max_burst(float *x, int n, int k, int *startend, float *bigmax) {
		const int numthreadsBlock = 1024;
		int numBlock = ( n + numthreadsBlock - 1)/numthreadsBlock;
		//declare arrays on cpu to store the results from the kernel
		float *maxarr = (float *)malloc(numBlock * sizeof(float));
  int *maxstart = (int *)malloc(numBlock * sizeof(int));
  int *maxend = (int *)malloc(numBlock * sizeof(int));

		// declare GPU memory pointers
		float *darr, * dmaxarr;
		int *dmaxstart, *dmaxend;
		hipMalloc((void **)&darr, n*sizeof(float));
		hipMalloc((void **)&dmaxarr, numBlock*sizeof(float));
		hipMalloc((void **)&dmaxstart, numBlock*sizeof(int));
		hipMalloc((void **)&dmaxend, numBlock*sizeof(int));

		//copy the input x to device
		hipMemcpy(darr, x, n*sizeof(float), hipMemcpyHostToDevice);
  // execution configuration
		dim3 dimGrid(numBlock,1);
		dim3 dimBlock(numthreadsBlock,1,1);
  //call the kernel
		max_each_block<<<dimGrid,dimBlock,(3*numthreadsBlock)*sizeof(float)>>>(dmaxarr,dmaxstart,dmaxend, darr, n, k);
		hipDeviceSynchronize();
		//copy the results from device to cpu
		hipMemcpy(maxarr, dmaxarr, numBlock*sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(maxstart, dmaxstart, numBlock*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(maxend, dmaxend, numBlock*sizeof(int), hipMemcpyDeviceToHost);
  //free memory on device
		hipFree(darr);
		hipFree(dmaxarr);
		hipFree(dmaxstart);
		hipFree(dmaxend);

  find_max_from_blocks( maxarr, maxstart, maxend,  numBlock,startend, bigmax);
  printf("burst start from %d end at %d; max-mean is %f\n", startend[0], startend[1], *bigmax);

}

void find_max_from_blocks(float *maxarr, int *maxstart, int *maxend, int numBlock,int *startend, float *bigmax) {
	*bigmax = 0;
	for (int i = 0; i < numBlock; i++) {
   if (*bigmax < maxarr[i]) {
      *bigmax = maxarr[i];
      startend[0] = maxstart[i];
     	startend[1] = maxend[i];
			}	
 }
}

void s_max_burst(float *arr, int n, int k) {
   float mymaxval = -1; 
   int perstart, perlen,perend, mystart, myend;
   float xbar;
   for (perstart = 0; perstart <= n-k; perstart++) {
     for (perlen = k; perlen <= n - perstart; perlen++) {
        perend = perstart + perlen -1; 
        xbar = mean(arr, perstart, perend);
     if (xbar > mymaxval) {
        mymaxval = xbar;
        mystart = perstart;
        myend = perend;
     }   
     }   
   }   
   printf("\nburst start from %d end %d, max-mean is %f\n", mystart, myend,mymaxval);
}

float mean(float *y, int s, int e){ 
  int i;  
  float tot =0; 
  for (i=s; i<=e; i++) tot += y[i];
  return tot / (e -s + 1); 
}
