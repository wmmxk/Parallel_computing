#include "hip/hip_runtime.h"

/* declare a 1d array and find the maximum of each chunk using reduce method.
	*chunksize must be an exponential of 2. chunksize is equal to the nubmer of threads in a block

	how to compile: 
	 nvcc shared_mem.cu
	how to run
  ./a.out n 
	 n 	is a number defining the length of the input array;	
  n is 600,000 or more, the results are not correct probably because there is not enough threads.

 The 1d array used for testing is a sequence from 0 to n-1.

	*/

#include <stdio.h>
#include <hip/hip_runtime.h>


float * serial_max_each_chunk(float maxarr[], float arr[], int chunkSize, int n);
__global__ void parallel_max_each_chunk(float *dmaxarr, float * darr, int chunkSize, int n);


int main(int argc, char **argv) {
//generate a 1d array
int n = atoi(argv[1]);
float *arr = (float*) malloc(n*sizeof(float));
int i;
for (i =0; i < n; i++) {
   arr[i] = (float)i/2.0f;			
}

const int chunkSize = 512;
//const int chunkSize = atoi(argv[2]);//chunkSize should be an exponential of 2
int numChunk = (n + chunkSize -1)/chunkSize;

float *maxarr = (float *)malloc(numChunk * sizeof(float));

// declare GPU memory pointers
float *darr, * dmaxarr;
hipMalloc((void **)&darr, n*sizeof(float));
hipMalloc((void **)&dmaxarr, numChunk*sizeof(float));
hipMemcpy(darr, arr, n*sizeof(float), hipMemcpyHostToDevice);

dim3 dimGrid(numChunk,1);
dim3 dimBlock(chunkSize,1,1);

parallel_max_each_chunk<<<dimGrid,dimBlock,chunkSize*sizeof(float)>>>(dmaxarr, darr, chunkSize,n);
hipDeviceSynchronize();
hipMemcpy(maxarr, dmaxarr, numChunk*sizeof(float), hipMemcpyDeviceToHost);

for (i=0; i < numChunk; i++) {
printf("%d maximum: %f\n",i,maxarr[i]);
}

// solution by a serial solution
float * smaxarr = (float *) malloc(numChunk * sizeof(float));
printf("\nserial solution\n");
serial_max_each_chunk(smaxarr, arr, chunkSize, n);

//compare two solutions
bool judge = true;
for (i=0; i < numChunk; i++) {
printf("%d maximum: %f\n",i,smaxarr[i]);
judge = judge && (smaxarr[i] == maxarr[i]);
}

printf("\n--------correct or wrong---------\n");
printf(judge ? "right\n": "wrong\n");

// check the exit state of CUDA code
hipError_t error = hipGetLastError();
if (error !=hipSuccess) {
  printf("CUDA error: %s\n", hipGetErrorString(error));
  exit(-1);
}

return 0;
}

//serial solution
float * serial_max_each_chunk(float maxarr[], float arr[], int chunkSize, int n) {
		int numChunk = (n + chunkSize - 1)/chunkSize; 
		int i,j;
  for (i = 0; i < numChunk; i++){
				 maxarr[i] = -3.0;
					for (j = i * chunkSize; j < (i+1)*chunkSize; j++) {
									if (j >= n) { break;
									} else { 
											if (maxarr[i] < arr[j]) { maxarr[i] = arr[j];}
									}
					}   
		}
		return maxarr;
}



__global__ void parallel_max_each_chunk(float *dmaxarr, float * darr, int chunkSize, int n) {
  int myId = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;

  extern __shared__ float sdata[];
  //copy each chunk to the shared memory for this block;
		sdata[tid] = myId >=n? 0:darr[myId];
  __syncthreads();


//do reduce on the chunk of array on the shared memory
  for (int s = blockDim.x/2; s > 0; s>>=1) {
     if (tid < s ) {
       sdata[tid]= sdata[tid +s]  > sdata[tid]? sdata[tid+s] : sdata[tid];
     }
     __syncthreads();
  }

// the maximum in this chunk is the sdata[0]
  if(tid == 0) {
  dmaxarr[blockIdx.x] = sdata[0];
  }
}

