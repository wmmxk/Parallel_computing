#include "hip/hip_runtime.h"
// n should be less than 10000 when k==3
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void parallel_max_each_chunk(float *dmaxarr, float * darr,  int n, int k);


int main(int argc, char **argv) {
				int n = atoi(argv[1]);
				int k = atoi(argv[2]); 
				
		//generate a 1d array
				float *arr = (float*) malloc(n*sizeof(float));
				int i;
				for (i = n; i > 0; i--) {
							arr[n-i] = (float)i;			
				}

		const int numthreadsBlock = 8;
		int numChunk = ( n + numthreadsBlock - 1)/numthreadsBlock;

		float *maxarr = (float *)malloc(numChunk * sizeof(float));

		// declare GPU memory pointers
				float *darr, * dmaxarr;
				hipMalloc((void **)&darr, n*sizeof(float));
				hipMalloc((void **)&dmaxarr, numChunk*sizeof(float));
				hipMemcpy(darr, arr, n*sizeof(float), hipMemcpyHostToDevice);

		dim3 dimGrid(numChunk,1);
		dim3 dimBlock(numthreadsBlock,1,1);

		parallel_max_each_chunk<<<dimGrid,dimBlock,(n+3*numthreadsBlock)*sizeof(float)>>>(dmaxarr, darr, n, k);
		hipDeviceSynchronize();
		hipMemcpy(maxarr, dmaxarr, numChunk*sizeof(float), hipMemcpyDeviceToHost);


  //truth
				float *smaxarr = (float *)malloc(numChunk*sizeof(float));
				for (i = 0; i < numChunk; i ++) {
        smaxarr[i] = i*numthreadsBlock + 1<=n? arr[i*numthreadsBlock + 1]:0;
				}


		//check the results
				bool judge = true;
				for (i=0; i < numChunk; i++) {
						printf("max of block  %d,  %f\n ", i, smaxarr[i]);
						judge = judge && (smaxarr[i] == maxarr[i]);
				}
				printf("\n--------correct or wrong---------\n");
				printf(judge ? "right\n": "wrong\n");

				// This is for developing: print out the 1d array

				printf("\n--------1d array---------\n");
    if ( n < 15) {
				for (i=0; i < n; i++) {
						printf("element  %d,  %f\n ", i, arr[i]);
				}
    }
		// check the exit state of CUDA code
		hipError_t error = hipGetLastError();
		if (error !=hipSuccess) {
				printf("CUDA error: %s\n", hipGetErrorString(error));
		}
		return 0;
}

__global__ void parallel_max_each_chunk(float *dmaxarr, float * darr, int n,int k) {
 	int i, tid = threadIdx.x;
		//copy the whole series to shared memory
  //always round up and if n is a multiple of blockDim.x no rounding
		int chunkSize = (n+blockDim.x-1)/blockDim.x;
  extern __shared__ float sdata[];
  for (i = 0; i < chunkSize; i++) {
			  if (tid * chunkSize + i <n)
     sdata[tid*chunkSize + i ] = darr[tid*chunkSize + i];
			}
  __syncthreads();

  // declare three array for the maximum found by each thread 
  extern __shared__ float mymaxvals[];
  extern __shared__ float mystartmaxes[];
  extern __shared__ float myendmaxes[];
		
		int perstart = threadIdx.x + blockDim.x * blockIdx.x;
		int perlen, perend;
		double xbar; // a temporay variable used when computing mean of subsequence

		if (perstart <= n-k) {
    for (perlen = k ; perlen <= n - perstart ; perlen++) {
				  perend = perstart + perlen - 1;
						//compute the mean of subsequence incrementally
      if (perlen ==k) {
							  xbar = 0;
									for ( i = perstart; i <= perend; i++) {
           xbar += sdata[i];     
									}
									xbar /= (perend - perstart + 1);
									mymaxvals[tid] = xbar;
						} else {
        xbar = ( (perlen-1) * xbar + sdata[perend] ) / perlen;
						}
						//update the mymaxvals[tid] if the next longer subsequence has a higher mean
						if (xbar > mymaxvals[tid]) {
         mymaxvals[tid] = xbar;
									mystartmaxes[tid] = perstart;
									myendmaxes[tid] = perend;
						}
				}
		} else {
    mymaxvals[tid] = 0;//initialize it the smallest number
		}
//  mymaxvals[tid] = sdata[tid]; 
		__syncthreads(); //sync to make sure each thread in this block has done with the for loop

  // get the highest among the mymaxvals using reduce
  for (int s = blockDim.x/2; s > 0; s>>=1) {
     if (tid < s ) {
       if(mymaxvals[tid+s] > mymaxvals[tid]) {
						   	mymaxvals[tid] = 	mymaxvals[tid+s]; 
          mystartmaxes[tid] = mystartmaxes[tid + s];
									 myendmaxes[tid] = myendmaxes[tid + s];	
							}	
     }
     __syncthreads();
  }
  // the maximum among the mymaxvals in this block
  if(tid == 0) {
  dmaxarr[blockIdx.x] = mymaxvals[0];
  }
}

