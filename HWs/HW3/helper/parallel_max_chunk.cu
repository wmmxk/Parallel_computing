#include "hip/hip_runtime.h"

/* declare a 1d array and find the maximum of each chunk using reduce method. No shared memory is used
	*
	*chunksize must be an exponential of 2
how to compile: nvcc para

when n is 600,000 or more, the results are not correct probably because there is not enough threads.

The 1d array used for testing is a sequence from 0 to n-1.

How to deal with the incomplete chunk:
 
     if (tid < s && myId < n) { //myId >=n the incomplete chunk is less than blockDim.x/2) 
						 float right_counterpart = (myId+s) >= n? 0:darr[myId+s]; //if the right_counterpart is missing, use 0

	*/

#include <stdio.h>
#include <hip/hip_runtime.h>


float * serial_max_each_chunk(float maxarr[], float arr[], int chunkSize, int n);
__global__ void parallel_max_each_chunk(float *dmaxarr, float * darr, int chunkSize, int n);


int main(int argc, char **argv) {
//generate a 1d array
int n = atoi(argv[1]);
float *arr = (float*) malloc(n*sizeof(float));
int i;
for (i =0; i < n; i++) {
   arr[i] = (float)i/2.0f;			
}

const int chunkSize = 512;
int numChunk = (n + chunkSize -1)/chunkSize;

float *maxarr = (float *)malloc(numChunk * sizeof(float));

// declare GPU memory pointers
float *darr, * dmaxarr;
hipMalloc((void **)&darr, n*sizeof(float));
hipMalloc((void **)&dmaxarr, numChunk*sizeof(float));
hipMemcpy(darr, arr, n*sizeof(float), hipMemcpyHostToDevice);

dim3 dimGrid(numChunk,1);
dim3 dimBlock(chunkSize,1,1);


parallel_max_each_chunk<<<dimGrid,dimBlock>>>(dmaxarr, darr, chunkSize,n);
hipDeviceSynchronize();
hipMemcpy(maxarr, dmaxarr, numChunk*sizeof(float), hipMemcpyDeviceToHost);



for (i=0; i < numChunk; i++) {
printf("%d maximum: %f\n",i,maxarr[i]);
}

float * smaxarr = (float *) malloc(numChunk * sizeof(float));
printf("\nserial solution\n");
serial_max_each_chunk(smaxarr, arr, chunkSize, n);

bool judge = true;
for (i=0; i < numChunk; i++) {
printf("%d maximum: %f\n",i,smaxarr[i]);
judge = judge && (smaxarr[i] == maxarr[i]);
}

printf("\n--------correct or wrong---------\n");
printf(judge ? "right\n": "wrong\n");



// check the exit state of CUDA code
hipError_t error = hipGetLastError();
if (error !=hipSuccess) {
  printf("CUDA error: %s\n", hipGetErrorString(error));
  exit(-1);
}






return 0;
}


float * serial_max_each_chunk(float maxarr[], float arr[], int chunkSize, int n) {
		int numChunk = (n + chunkSize - 1)/chunkSize; 
		int i,j;
  for (i = 0; i < numChunk; i++){
				 maxarr[i] = -3.0;
					for (j = i * chunkSize; j < (i+1)*chunkSize; j++) {
									if (j >= n) { break;
									} else { 
											if (maxarr[i] < arr[j]) { maxarr[i] = arr[j];}
									}
					}   
		}
		return maxarr;
}



__global__ void parallel_max_each_chunk(float *dmaxarr, float * darr, int chunkSize, int n) {
  int myId = blockIdx.x * blockDim.x + threadIdx.x;
  int tid = threadIdx.x;

  for (int s = blockDim.x/2; s > 0; s>>=1) {
     if (tid < s && myId < n) { //myId >=n the incomplete chunk is less than blockDim.x/2) 
						 float right_counterpart = (myId+s) >= n? 0:darr[myId+s]; //if the right_counterpart is missing, use 0
       darr[myId]= right_counterpart > darr[myId]? right_counterpart : darr[myId];
     }
     __syncthreads();
  }
  if(tid == 0) {
  dmaxarr[blockIdx.x] = darr[myId];
  }
}

