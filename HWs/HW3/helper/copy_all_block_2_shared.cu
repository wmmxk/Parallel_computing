#include "hip/hip_runtime.h"
/* declare a 1d array and copy it to each block
	  each thread find the maximum of the 1d array. This will be replaced by the inner loop in HW 
	  use reduce sum up the mymaximum found by each thread
	  finally each block return a sum of maximum.		
	
  parallel_max_each_chunk<<<dimGrid,dimBlock,(n+numthreadsBlock)*sizeof(float)>>>(dmaxarr, darr, n);
     It seems you can request less memory
	*/
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void parallel_max_each_chunk(float *dmaxarr, float * darr,  int n);


int main(int argc, char **argv) {
//generate a 1d array
int n = atoi(argv[1]);
float *arr = (float*) malloc(n*sizeof(float));
int i;
for (i =0; i < n; i++) {
   arr[i] = (float)i/2.0f;			
}

const int numthreadsBlock = 8;
int numChunk = atoi(argv[2]);

float *maxarr = (float *)malloc(numChunk * sizeof(float));

// declare GPU memory pointers
float *darr, * dmaxarr;
hipMalloc((void **)&darr, n*sizeof(float));
hipMalloc((void **)&dmaxarr, numChunk*sizeof(float));
hipMemcpy(darr, arr, n*sizeof(float), hipMemcpyHostToDevice);

dim3 dimGrid(numChunk,1);
dim3 dimBlock(numthreadsBlock,1,1);

parallel_max_each_chunk<<<dimGrid,dimBlock,(n+numthreadsBlock)*sizeof(float)>>>(dmaxarr, darr, n);
hipDeviceSynchronize();
hipMemcpy(maxarr, dmaxarr, numChunk*sizeof(float), hipMemcpyDeviceToHost);


//check the results
bool judge = true;
for (i=0; i < numChunk; i++) {
printf("%d sum of max %f\n ", i, maxarr[i]);
judge = judge && ( (n-1)*numthreadsBlock/2.0 == maxarr[i]);
}
printf("\n--------correct or wrong---------\n");
printf(judge ? "right\n": "wrong\n");

// check the exit state of CUDA code
hipError_t error = hipGetLastError();
if (error !=hipSuccess) {
  printf("CUDA error: %s\n", hipGetErrorString(error));
}


for (i=0; i < n; i++) {
printf("%d element  %f\n ", i, arr[i]);
}

return 0;
}

__global__ void parallel_max_each_chunk(float *dmaxarr, float * darr, int n) {
  int tid = threadIdx.x;
  int j;
		int chunkSize = (n+blockDim.x-1)/blockDim.x;

  extern __shared__ float sdata[];
  for (j = 0; j < chunkSize; j++) {
			  if (tid * chunkSize +j <n)
     sdata[tid*chunkSize + j ] = darr[tid*chunkSize + j];
			}
  __syncthreads();


 // each thread find the maximum of the sdata 
  extern __shared__ float mymaxval[];

		int mymax = 0;
		for ( j =0; j < n; j++)
		{
    if (mymax < sdata[j]) { mymax = sdata[j];}
		}
  mymaxval[tid] = mymax;

//do reduce on the chunk of array on the shared memory
  for (int s = blockDim.x/2; s > 0; s>>=1) {
     if (tid < s ) {
       mymaxval[tid] += mymaxval[tid+s]; 
     }
     __syncthreads();
  }

// the sum of the maximum found by each thread
  if(tid == 0) {
  dmaxarr[blockIdx.x] = mymaxval[0];
  }
}

